#include "hip/hip_runtime.h"
/*

author: Subed lamichhane
cuSolver Implementation
inputs: input matrix, host eigen values matrix and host eigen vector matrixa and size of matrix(n_row=n_col=size)
refrence: cuSolver official site by NVIDIA

*/



#include <hip/hip_runtime.h>
#include <hipsolver.h>
//#include "kernel.cu"
//#include "support.cu"


int  solver_eigen (double *C_hd, double *W,double *V,int size )
{
    hipsolverHandle_t cusolverH = NULL;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    const int m = size;
    const int lda = m;

    double *d_A = NULL;
    double *d_W = NULL;
    int *devInfo = NULL;
    double *d_work = NULL;
    int  lwork = 0;

    int info_gpu = 0;

//    printf("A = (matlab base-1)\n");
//    printMatrix(m, m, A, lda, "A");
//    printf("=====\n");


//call eigenvalue solver
 
// step 1: create cusolver/cublas handle
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

// step 2: copy A and B to device
    cudaStat1 = hipMalloc ((void**)&d_A, sizeof(double) * lda * m);
    cudaStat2 = hipMalloc ((void**)&d_W, sizeof(double) * m);
    cudaStat3 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    cudaStat1 = hipMemcpy(d_A, C_hd, sizeof(double) * lda * m, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);

// step 3: query working space of syevd
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    cusolver_status = hipsolverDnDsyevd_bufferSize(
        cusolverH,
        jobz,
        uplo,
        m,
        d_A,
        lda,
        d_W,
        &lwork);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

// step 4: compute spectrum
    cusolver_status = hipsolverDnDsyevd(
        cusolverH,
        jobz,
        uplo,
        m,
        d_A,
        lda,
        d_W,
        d_work,
        lwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(W, d_W, sizeof(double)*m, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(V, d_A, sizeof(double)*lda*m, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);


//check the result
 
    printf("after syevd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);

    printf("eigenvalue = (matlab base-1), ascending order\n");
    for(int i = 0 ; i < m ; i++){
        printf("W[%d] = %E\n", i+1, W[i]);
    }

    printf("V = (matlab base-1)\n");
    printMatrix(m, m, V, lda, "V");
    printf("=====\n");

// step 4: check eigenvalues
//  double lambda_sup = 0;
//  for(int i = 0 ; i < m ; i++){
//      double error = fabs( lambda[i] - W[i]);
//      lambda_sup = (lambda_sup > error)? lambda_sup : error;
//  }
//  printf("|lambda - W| = %E\n", lambda_sup);

// free resources
    if (d_A    ) hipFree(d_A);
    if (d_W    ) hipFree(d_W);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);

    if (cusolverH) hipsolverDnDestroy(cusolverH);

    hipDeviceReset();

    return 0;
    
}


