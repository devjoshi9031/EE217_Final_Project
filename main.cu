#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include "kernel.cu"
#include "support.cu"
#include "getEigen.cu"
#include "matMul.cu"
#include "transpose.cu"


//Matrix Print ; not working as expected right now
void printMatrixFloat(int m, int n, const float*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        printf("\n");
        for(int col = 0 ; col < n ; col++){
            float Areg = A[row*lda + col];
            printf("%s(%d,%d) = %f\t", name, row+1, col+1, Areg);
        }
    }
}



int main(int argc, char*argv[])
{

   float *A_h,*B_h,*C_h,*D_h;
//   float *A_d, *B_d, *C_d,*D_d;
    size_t A_sz, B_sz, C_sz,D_sz;
    unsigned matArow, matAcol;
    unsigned matBrow, matBcol;
//  dim3 dim_grid, dim_block;
//  hipError_t cuda_ret;
    double *C_hd,*D_hd;

    int mm,n,k;

    matArow=2;
    matAcol=3;
    matBrow=matAcol;
    matBcol=matArow;

    A_sz = matArow*matAcol;
    B_sz = matBrow*matBcol;
    C_sz = matArow*matBcol;
    D_sz = matBrow*matAcol;
 
// toy matrices initialization
   A_h = (float*) malloc( sizeof(float)*A_sz );


    A_h[0] =1.0;
    A_h[1] =2.0;
    A_h[2] =3.0;
    A_h[3] =2.0;
    A_h[4] =3.0;
    A_h[5] =4.0;

// B is A transpose
    B_h = (float*) malloc( sizeof(float)*B_sz );
    

//    B_h[0] =1.0;
//    B_h[1] =2.0;
//    B_h[2] =2.0;
//    B_h[3] =3.0;
//    B_h[4] =3.0;
//    B_h[5] =4.0;

    basicTransp(matArow,matAcol,A_h,B_h);

// C_hd and D_hd are double version of C_h and D_h

    C_h = (float*) malloc( sizeof(float)*C_sz );
    C_hd = (double*) malloc( sizeof(double)*C_sz );

    D_h = (float*) malloc( sizeof(float)*D_sz );
    D_hd = (double*) malloc( sizeof(double)*D_sz );


    printMatrixFloat(matArow, matAcol, A_h, matAcol, "A");
    printf("\n");
    printMatrixFloat(matBrow, matBcol, B_h, matBcol, "B");


    mm=matArow;
    n=matAcol;
    k=matBcol;

// matric multiplication for AA' or vice versa
    matrix_multiply(A_h,B_h,C_h,mm,n,n,k);
  
   
    mm=matAcol;
    n=matArow;
    k=matBrow;

// matrix multiplication for A'A
    matrix_multiply(B_h,A_h,D_h,mm,n,n,k);


    for (int i=0;i<C_sz;i++){C_hd[i]=(double)C_h[i];}
    for (int i=0;i<D_sz;i++){D_hd[i]=(double)D_h[i];}

    printf("\nresulting matrix AA'  in double format is \n");
    printMatrix(matArow, matBcol, C_hd, matArow, "C");     

    printf("\nresulting matrix A'A  in double format is \n");
    printMatrix(matBrow, matAcol, D_hd, matBrow, "D");    

    
// cusolver call starts

    int  m= matArow;
    int lda=m;
//    double lambda[m] = {22,2};
    double V[lda*m]; // eigenvectors
    double W[m]; // eigenvalues
   
// C_hd is the matrix input, W gives diagonal eigen value matrix and V gives right eigen vectors
    solver_eigen(C_hd,W,V,m);

    return 0;
}


