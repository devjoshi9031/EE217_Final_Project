/*
matMul.cu
author : Subed Lamichhane
Matrix multiplication
inputs: host A,B,C, n_rowA,n_colA, n_rowB, n_col_B
*/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>


int matrix_multiply(float *A_h, float *B_h, float *C_h, int matArow,int matAcol,int matBrow, int matBcol)
{

  
    float *A_d, *B_d, *C_d;
    size_t A_sz, B_sz, C_sz;

    dim3 dim_grid, dim_block;
    hipError_t cuda_ret;

    A_sz = matArow*matAcol;
    B_sz = matBrow*matBcol;
    C_sz = matArow*matBcol;


// Allocate device variables ----------------------------------------------

//    printf("Allocating device variables..."); 
    fflush(stdout);
//    startTime(&timer);

/*************************************************************************/

//INSERT CODE HERE

    hipMalloc((void **) &A_d, A_sz*sizeof(float));
    hipMalloc((void **) &B_d, B_sz*sizeof(float));
    hipMalloc((void **) &C_d, C_sz*sizeof(float));


/*************************************************************************/

    hipDeviceSynchronize();


// Copy host variables to device ------------------------------------------
//    printf("Copying data from host to device..."); 
    fflush(stdout);
//  startTime(&timer);

/*************************************************************************/
//INSERT CODE HERE

    hipMemcpy(A_d, A_h, A_sz*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, B_sz*sizeof(float), hipMemcpyHostToDevice);

/*************************************************************************/

    hipDeviceSynchronize();
    
// Launch kernel using standard sgemm interface ---------------------------
//    printf("Launching kernel..."); 
    fflush(stdout);
//    startTime(&timer);


    basicSgemm(matArow, matBrow, matBcol, A_d, B_d, C_d);

  
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("Unable to launch kernel");
//    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

// Copy device variables from host ----------------------------------------
//    printf("Copying data from device to host..."); 
    fflush(stdout);
//    startTime(&timer);

/*************************************************************************/
//INSERT CODE HERE
    hipMemcpy(C_h, C_d, C_sz*sizeof(float), hipMemcpyDeviceToHost);

//    hipDeviceSynchronize();

//    hipMemcpy(D_h, D_d, C_sz*sizeof(float), hipMemcpyDeviceToHost);
/*************************************************************************/

//    printf("C_h %f", C_h);
    hipDeviceSynchronize();



   return 0;
}


