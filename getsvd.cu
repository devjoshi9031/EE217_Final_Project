#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h> 
#include <math.h>
//#include "support.cu"
//#define DEBUG

__global__ void svd_kernel(double *A, double *V, double *U, double *sig,int m, int n){

   __shared__ double N[16][1];
   __shared__ double M[16][16];
  
//    unsigned int x = blockIdx.x * BLOCK_DIM + threadIdx.x;
//    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;

    unsigned int tx=threadIdx.x;
    unsigned int ty=threadIdx.y;
    unsigned int bx=blockIdx.x;
    unsigned int by=blockIdx.y;

    int row= by*blockDim.y+ty;
    int col= bx*blockDim.x+tx;
    
    for (int i=0; i<n;i++){
      double Pvalue=0.0;
      if (row<m && col<n){
		M[ty][tx]=A[(row*n+col)];
	}
	else{
		M[ty][tx]=0.0;
        }
	if (row<m){
		N[ty][i]=V[row*n+i] ;
	}
	else{
		N[ty][tx]=0.0;
	}  
       __syncthreads();

      if (row<m && col<1){
		for (int j=0;j<n;++i){
			Pvalue+=M[ty][j]*N[j][tx]/sig[j];

		}
       
	}
     
      __syncthreads();

       if (row<m){
          U[row*n+i]=Pvalue;
          }
      __syncthreads();

}


}

int rank=0;
void SVD(double *V1, double *W1, double *V2, double *U, double *Sig, double *VT,int m,int nn ){

     printf("inside SVD func");

//    float * UTr;
    
//    float *U2f;
    double *V;
    double *sig;
//    UTr=(float*)malloc(sizeof(float)*m*m);
//    U2f=(float*)malloc(sizeof(float)*m*m);
    V=(double*)malloc(sizeof(double)*nn*nn);
//    sig=(double*)malloc(sizeof(double)*nn);

    int k=nn-1;
    int l=0;
    for (int i=0; i<m; i++){
        for(int j=0;j<nn;j++){
            if (i==j&& W1[k]>0.0){
                Sig[i*nn+j]=sqrt(W1[k]);
                sig[l]=Sig[i*nn+j];
                k=k-1;
                l=l+1;
                if (Sig[i*nn+j]>0.0){rank=rank+1;}    
         }
             else{Sig[i*nn+j]=0.0;}
         }
    }
    
    rank=l;
//    for (int i=0;i<nn;i++){printf("\nsig(%d):%f\t",i,sig[i]);}
//    for (int i=0;i<m*m;i++){U2f[i]=(float)V1[i];}
//    basicTransp(m,m,U2f,UTr);
//    for (int i=0;i<m*m;i++){Ud[i]=(double)UTr[i];}

  
    k=0;
    for (int i=0;i<m;i++){
        for (int j=m-1;j>=0;j--){
            U[i*m+j]=V1[i+m*k];
            k=k+1;
        }
        k=0;
    }
//
//
//    k=0;
//    
//    for (int i=0;i<nn*nn;i++){V2f[i]=(float)V2[i];}
//    basicTransp(nn,nn,V2f,VTd);
//    for (int i=0;i<nn*nn;i++){VTr[i]=(double)VTd[i];}

//    printMatrix(nn,nn,VTr,nn,"VTr");


    


    for (int i=nn-1;i>=0;i--){
        for (int j=0;j<nn;j++){
            VT[i*nn+j]=V2[k*nn+j];
            V[i*nn+j]=V2[i+nn*j];
       }
        k=k+1;
    }
     
//    int gridx,gridy;
//
//    gridx=ceil(nn/16.0);
//    gridy=ceil(m/16.0);
//
//    dim3 DimGrid(gridx,gridy,1); 
//    dim3 DimBlock(16,16,1);
//
//    svd_kernel<<<DimGrid,DimBlock>>>(A,V,U,sig,m,nn);

//    free(UTr);free(U2f);free(Ud);

}  







void compressed(double *U, double *Sig, double *VT, double *Mc,int m, int nn,int r){


  float *Uf;
  float *Sigf;
  float *VTf;
  float *Mf;
  float *Mtf;
  r=rank;
  Uf=(float*)malloc(sizeof(float)*m*m);
  Sigf=(float*)malloc(sizeof(float)*m*nn); 
  VTf=(float*)malloc(sizeof(float)*nn*nn);
  Mf=(float*)malloc(sizeof(float)*m*nn);
  Mtf=(float*)malloc(sizeof(float)*r*nn); 
//  float Mcf[r*r];

  
//  for(int i=0;i<m*m;i++){Uf[i]=(float)U[i];}

//  for(int i=0;i<m*nn;i++){Sigf[i]=(float)Sig[i];} 
//  for(int i=0;i<nn*nn;i++){VTf[i]=(float)VT[i];}
//  for(int i=0;i<m*nn;i++){Mf[i]=(float)M[i];}
//  for(int i=0;i<m*nn;i++){Mcf[i]=(float)Mc[i];}

  for (int i=0;i<m;i++){
     for(int j=0;j<r;j++){
        Uf[i*r+j]=(float)U[i*r+j+(m-r)*i];
     }
  }

  for (int i=0;i<r;i++){
     for(int j=0;j<nn;j++){
        VTf[i*nn+j]=(float)VT[i*nn+j];
     }
  }
 
  for (int i=0;i<r;i++){
     for(int j=0;j<r;j++){

        Sigf[i*r+j]=(float)Sig[i*nn+j];
     }
  }

//  #ifdef DEBUG

  printf("\nrank:  %d", r);
//
  printf("\n");
  printMatrixFloat(m,r,Uf,r,"Uf");
//
  printf("\n");
  printMatrixFloat(r,r,Sigf,r,"Sigf");
//
  printf("\n");
  printMatrixFloat(r,nn,VTf,nn,"VTf");
//  #endif 


  matrix_multiply(Sigf,VTf,Mtf,r,r,r,nn);
 
//  #ifdef DEBUG
//  printf("\n");
//  printMatrixFloat(r,nn,Mtf,nn,"Mtf");
//  #endif
 
  matrix_multiply(Uf,Mtf,Mf,m,r,r,nn);

//  #ifdef DEBUG 
//  printf("\n");
//  printMatrixFloat(m,nn,Mf,nn,"Mf");
//  #endif

  for (int i=0;i<m*nn;i++){Mc[i]=(double)Mf[i];}
  
  free(Uf);free(Sigf);free(VTf);free(Mf);free(Mtf);  

}


float rmse(double *M1, int row, int col){

  float square=0;
  float sum=0;
  float mean=0;
  float rms=0;

  for(int i=0; i<row;i++){
     for(int j=0;j<col;j++){
           square=M1[i*col+j]*M1[i*col+j];
//           printf("\n%f",square);
           sum=sum+square;
     }

   }

//   printf("\n%f",sum);
   mean=sum/(row*col);
//   printf("\nmean:%f",mean);
   rms=sqrt(mean);
//   printf("\nrms: %f",rms);
   return rms; 

}
   
