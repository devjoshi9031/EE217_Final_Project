#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>

#include "support.h"

//void verify(float *A, float *B, float *C, unsigned int n) {
//
//  const float relativeTolerance = 1e-2;
//
//  for(int i = 0; i < n; ++i) {
//      float sum = A[i]+B[i];
//      printf("\t%d:%f/%f",i,sum,C[i]);
//      float relativeError = (sum - C[i])/sum;
//      if (relativeError > relativeTolerance
//        || relativeError < -relativeTolerance) {
//        printf("\nTEST FAILED\n\n");
//        exit(0);
//      }
//  }
//  printf("\nTEST PASSED\n\n");
//
//}

void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer) {
    return ((float) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}


void printMatrix(int m, int n, const double*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        printf("\n");
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row*lda+ col];
            printf("%s(%d,%d) = %f\t", name, row+1, col+1, Areg);
        }
    }
}

