
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_DIM 16
#define BLOCK_DIM 16



__global__ void myTransp(float *odata, float *idata, int width, int height)
{
	__shared__ float block[BLOCK_DIM][BLOCK_DIM+1];
	
	// read the matrix tile into shared memory
        // load one element per thread from device memory (idata) and store it
        // in transposed order in block[][]
	unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
	unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
	if((xIndex < width) && (yIndex < height))
	{
		unsigned int index_in = yIndex * width + xIndex;
		block[threadIdx.y][threadIdx.x] = idata[index_in];
	}

        // synchronise to ensure all writes to block[][] have completed
	__syncthreads();

	// write the transposed matrix tile to global memory (odata) in linear order
	xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
	yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
	if((xIndex < height) && (yIndex < width))
	{
		unsigned int index_out = yIndex * height + xIndex;
		odata[index_out] = block[threadIdx.x][threadIdx.y];
	}
}


void basicTransp(int k, int m,float *A, float *B){

    float *A_d, *B_d ;
    size_t A_sz, B_sz;

//    dim3 dim_grid, dim_block;
    hipError_t cuda_ret;

    A_sz = m*k;
    B_sz = k*m;
    int gridx,gridy;

    fflush(stdout);
    
    hipMalloc((void **) &A_d, A_sz*sizeof(float));
    hipMalloc((void **) &B_d, B_sz*sizeof(float));

    hipDeviceSynchronize();

    hipMemcpy(A_d, A, A_sz*sizeof(float), hipMemcpyHostToDevice);
//    cudaMemcpy(B_d, B_h, B_sz*sizeof(float), cudaMemcpyHostToDevice);

    hipDeviceSynchronize();

    gridx=ceil(m/16.0);
    gridy=ceil(k/16.0);

//    printf("gridx : %d , gridy: %d",gridx,gridy);

    dim3 DimGrid(gridx,gridy,1); 
    dim3 DimBlock(16,16,1);


// Invoke CUDA kernel -----------------------------------------------------

    myTransp<<<DimGrid,DimBlock>>>(B_d,A_d,m,k);
	
   
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("Unable to launch kernel");

    fflush(stdout);
    
    hipMemcpy(B, B_d, B_sz*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
   
     

 
}


