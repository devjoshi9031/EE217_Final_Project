
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE

     __shared__  float M[TILE_SIZE][TILE_SIZE];
     __shared__  float N[TILE_SIZE][TILE_SIZE];

     unsigned int tx=threadIdx.x;
     unsigned int ty=threadIdx.y;
     unsigned int bx=blockIdx.x;
     unsigned int by=blockIdx.y;

     int row= by*blockDim.y+ty;
     int col= bx*blockDim.x+tx;

     float Pvalue=0.0;
     int phases;
     phases=(TILE_SIZE+n-1)/TILE_SIZE;  
     for (int p=0;p<phases;p++){
//        Pvalue=0;
	if (row<m && (p*TILE_SIZE+tx)<n){
		M[ty][tx]=A[(row*n+p*TILE_SIZE+tx)];
	}
	else{
		M[ty][tx]=0.0;
        }
	if ((p*TILE_SIZE+ty)<n && col<k){
		N[ty][tx]= B[(p*TILE_SIZE+ty)*k+col];
	}
	else{
		N[ty][tx]=0.0;
	}
	__syncthreads();

	if (row<m && col<k){
		for (int i=0;i<TILE_SIZE;++i){
			Pvalue+=M[ty][i]*N[i][tx];
		}
	}
	__syncthreads();

    }
    if (row< m && col< k){
	C[row*k+col]=Pvalue;
    }
     
        
    /*************************************************************************/
}

void basicSgemm(int m, int n, int k, const float *A, const float *B, float *C){
    // Initialize thread block and kernel grid dimensions ---------------------

//    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE
//    int max=0,max_mn=0;

    int gridx,gridy;
//    max_mn=(m>n)? m:n;
//    max=(max_mn>k)? max_mn:k;
//    dim3 dimGrid.x(ceil(max/TILE_SIZE));
 //   dim3 dimGrid.y(ceil(max/TILE_SIZE));
    
    gridx=ceil(k/16.0);
    gridy=ceil(m/16.0);
 
//    printf("k  %d, gridx %d  gridy %d    ",k,gridx,gridy);
    dim3 DimGrid(gridx,gridy,1); 
    dim3 DimBlock(16,16,1);


    /*************************************************************************/

    // Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE

     mysgemm<<<DimGrid,DimBlock>>>(m,n,k,A,B,C);
	
    /*************************************************************************/
}


