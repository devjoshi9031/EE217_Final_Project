#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
//#include <hip/hip_runtime.h>
//#include <hipsolver.h>
//#include "kernel.cu"
//#include "support.cu"
//#include "getEigen.cu"
//#include "matMul.cu"
//#include "transpose.cu"
//#include <math.h>
//#include "getsvd.cu"


////Matrix Print ; not working as expected right now
//void printMatrixFloat(int m, int n, const float*A, int lda, const char* name)
//{
//    for(int row = 0 ; row < m ; row++){
//        printf("\n");
//        for(int col = 0 ; col < n ; col++){
//            float Areg = A[row*lda + col];
//            printf("%s(%d,%d) = %f\t", name, row+1, col+1, Areg);
//        }
//    }
//}


//#define DEBUG
void  svdCalc(float *M,int row,int col,double *U, double *Sig, double *VT)
{

   float *A_h,*B_h,*C_h,*D_h;
//   float *A_d, *B_d, *C_d,*D_d;
    size_t A_sz, B_sz, C_sz,D_sz;
    unsigned matArow, matAcol;
    unsigned matBrow, matBcol;
//  dim3 dim_grid, dim_block;
//  hipError_t cuda_ret;
    double *C_hd,*D_hd;

    int mm,n,k;

    matArow=row;
    matAcol=col;
    matBrow=matAcol;
    matBcol=matArow;

    A_sz = matArow*matAcol;
    B_sz = matBrow*matBcol;
    C_sz = matArow*matBcol;
    D_sz = matBrow*matAcol;
 
// toy matrices initialization
   A_h = (float*) malloc( sizeof(float)*A_sz );
   for(int i=0;i<A_sz;i++){A_h[i]=M[i];}

//    A_h[0] =1.0;
//    A_h[1] =2.0;
//    A_h[2] =3.0;
//    A_h[3] =2.0;
//    A_h[4] =3.0;
//    A_h[5] =4.0;

// B is A transpose
    B_h = (float*) malloc( sizeof(float)*B_sz );
    

//    B_h[0] =1.0;
//    B_h[1] =2.0;
//    B_h[2] =2.0;
//    B_h[3] =3.0;
//    B_h[4] =3.0;
//    B_h[5] =4.0;

    basicTransp(matArow,matAcol,A_h,B_h);

// C_hd and D_hd are double version of C_h and D_h

    C_h = (float*) malloc( sizeof(float)*C_sz );
    C_hd = (double*) malloc( sizeof(double)*C_sz );

    D_h = (float*) malloc( sizeof(float)*D_sz );
    D_hd = (double*) malloc( sizeof(double)*D_sz );

    #ifdef DEBUG
//    printMatrixFloat(matArow, matAcol, A_h, matAcol, "A");
//    printf("\n");
//    printMatrixFloat(matBrow, matBcol, B_h, matBcol, "B");
    #endif

    mm=matArow;
    n=matAcol;
    k=matBcol;

// matric multiplication for AA' or vice versa
    matrix_multiply(A_h,B_h,C_h,mm,n,n,k);
  
   
    mm=matAcol;
    n=matArow;
    k=matBrow;

// matrix multiplication for A'A
    matrix_multiply(B_h,A_h,D_h,mm,n,n,k);


    for (int i=0;i<C_sz;i++){C_hd[i]=(double)C_h[i];}
    for (int i=0;i<D_sz;i++){D_hd[i]=(double)D_h[i];}

    #ifdef DEBUG
    printf("\nresulting matrix AA'  in double format is \n");
//    printMatrix(matArow, matBcol, C_hd, matArow, "C");     

    printf("\nresulting matrix A'A  in double format is \n");
//    printMatrix(matBrow, matAcol, D_hd, matBrow, "D");    
    #endif
    
// cusolver call starts

    int  m= matArow;
    int lda=m;
//    double lambda[m] = {22,2};
    double *V1; // eigenvectors
    double *W1; // eigenvalues
   
    V1=(double*)malloc(sizeof(double)*m*lda);
    W1=(double*)malloc(sizeof(double)*m);


// C_hd is the matrix input, W gives diagonal eigen value matrix and V gives right eigen vectors
//    #ifdef DEBUG
    printf("\n Eigen decomposition for AA'\n");

    solver_eigen(C_hd,W1,V1,m);
  
    int nn=matAcol;
    lda=nn;
    double *V2; // eigenvectors
    double *W2; // eigenvalues

    V2=(double*)malloc(sizeof(double)*lda*nn);
    W2=(double*)malloc(sizeof(double)*nn);


// C_hd is the matrix input, W gives diagonal eigen value matrix and V gives right eigen vectors
    printf("\n Eigen decomposition for A'A \n");
    solver_eigen(D_hd,W2,V2,nn);
   
//    double S[m*nn];
//    k=nn-1;
//    for (int i=0; i<m; i++){
//        for(int j=0;j<nn;j++){
//            if (i==j){
//                S[i*nn+j]=sqrt(W2[k]);
//                k=k-1;
//            } 
//        }   
//    }
//   
////   printMatrix(1,nn,W2,nn,"W2"); 
//   printf("\n the singular values matrix is \n");
//   printMatrix(m,nn,S,nn,"S");
//   printf("\n");

   printf("\n Extrcting USigVT \n");

 //  double Sig[m*nn];
 //  double U[m*m];
 //  double VT[nn*nn];

//   double A[A_sz];
 
//   for(int i=0;i<A_sz;i++){A[i]=(double)M[i];}

//    printf("A calculated");

   SVD(V1,W2,V2,U,Sig,VT,m,nn);

//   #ifdef DEBUG
   printf("\n Left  Singular Matrix U\n");
   printMatrix(m,m,U,m,"U");

   printf("\n Singular values  Matrix Sig \n");
   printMatrix(m,nn,Sig,nn,"Sig");

   printf("\n Right  Singular Matrix U\n");
   printMatrix(nn,nn,VT,nn,"VT");
//   #endif
   
   free(A_h);free(B_h);free(C_h);free(D_h);
   free(W1);free(V1);free(W2);free(V2);

 
//    return 0;
}


